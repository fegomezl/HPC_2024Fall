#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h> 
#include <hip/hip_runtime.h>

#define N 1024

struct timeval startTime;
struct timeval finishTime;
double timeIntervalLength;

__global__ void CUDA_rk4_0(const double h, const double pow[], const double c[], const double y[], double k[]){
	int my_tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (0 < my_tid && my_tid < N){
		int j;
		double my_k = pow[my_tid];
		for (j = 0; j < N; j++)
			my_k -= c[my_tid*N+j]*y[j];
		my_k *= h;
		k[my_tid] = my_k;
	}
}

__global__ void CUDA_rk4_1(const double h, const double pow[], const double c[], const double y[], const double k_old[], double k_new[]){
	int my_tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (0 < my_tid && my_tid < N){
		int j;
		double my_k = pow[my_tid];
	        for (j = 0; j < N; j++)
			my_k -= c[my_tid*N+j]*(y[j]+0.5*k_old[j]);
		my_k *= h;
		k_new[my_tid] = my_k;
	}
}

__global__ void CUDA_rk4_2(const double h, const double pow[], const double c[], const double y[], 
						   const double k1[], const double k2[], const double k3[], double k4[], double yout[]){
	int my_tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (0 < my_tid && my_tid < N){
		int j;
		double my_k = pow[my_tid];
	        for (j = 0; j < N; j++)
			my_k -= c[my_tid*N+j]*(y[j]+k3[j]);
		my_k *= h;
		k4[my_tid] = my_k;

		yout[my_tid] = y[my_tid] + (k1[my_tid] + 2*k2[my_tid] + 2*k3[my_tid] + k4[my_tid])/6.0;
	}
}

int main(int argc, char* argv[]){

	// Define variables
	int i, j;
	double h;
	double totalSum;
	double*  y;
	double*  k1;
	double*  k2;
	double*  k3;
	double*  k4;
	double*  pow;
	double*  yout;
	double*  c;

	// Allocate arrays
	hipMallocManaged(&y, N*sizeof(double));
	hipMallocManaged(&k1, N*sizeof(double));
	hipMallocManaged(&k2, N*sizeof(double));
	hipMallocManaged(&k3, N*sizeof(double));
	hipMallocManaged(&k4, N*sizeof(double));
	hipMallocManaged(&pow, N*sizeof(double));
	hipMallocManaged(&yout, N*sizeof(double));
	hipMallocManaged(&c, N*N*sizeof(double));

	// Initialize variables
	h = 0.3154;
	totalSum = 0.0;
	for (i = 0; i < N; i++){
		y[i] = i*i;
		pow[i] = 2*i;
		for (j = 0; j < N; j++)
			c[i*N+j] = i*i+j;
	}
	
	// Get the start time
	gettimeofday(&startTime, NULL);

	CUDA_rk4_0<<<1024,1024>>>(h, pow, c, y, k1);
	hipDeviceSynchronize();	

	CUDA_rk4_1<<<1024,1024>>>(h, pow, c, y, k1, k2);
	hipDeviceSynchronize();	

	CUDA_rk4_1<<<1024,1024>>>(h, pow, c, y, k2, k3);
	hipDeviceSynchronize();	

	CUDA_rk4_2<<<1024,1024>>>(h, pow, c, y, k1, k2, k3, k4, yout);
	hipDeviceSynchronize();

	// Get the end time
	gettimeofday(&finishTime, NULL);

	// Check results
	for (i = 0; i < N; i++)
		totalSum += yout[i];
	printf("\n\ntotalSum=%g\n\n",totalSum);
	
	// Calculate the interval length 
	timeIntervalLength = (double)(finishTime.tv_sec-startTime.tv_sec) * 1000000 
	                   + (double)(finishTime.tv_usec-startTime.tv_usec);
	timeIntervalLength = timeIntervalLength/1000;

	// Print the interval lenght
	printf("Interval length: %g msec.\n", timeIntervalLength);

	// Free memory
	hipFree(y);
	hipFree(k1);
	hipFree(k2);
	hipFree(k3);
	hipFree(k4);
	hipFree(pow);
	hipFree(yout);
	hipFree(c);

	return 0;
}
