#include <stdio.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

// System size
#define		N	1048576

// Structures for profilling
struct timeval startTime;
struct timeval finishTime;
double timeIntervalLength;

// Each thread sums its contribution to the global sum
__global__ void CUDA_rect(const double h, double* area){
	int my_tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (my_tid < N)
		atomicAdd(area, cos(my_tid*h));
}

int main(int argc, char* argv[]){

	// Define variables
	double h;
	double *area;

	// Allocate variables
	hipMallocManaged(&area, sizeof(double));
	
	// Initialize variables
	h = 10.0/N;
	*area = 0.0;

	// Get the start time
	gettimeofday(&startTime, NULL);

	// Run device kernel and syncronize device
	CUDA_rect<<<N/1024,1024>>>(h, area);
	hipDeviceSynchronize();
	*area = h*(*area);

	// Get the end time
	gettimeofday(&finishTime, NULL);
		
	// Check results
	if (argc < 2)
		printf("Result : %.2lf \n", *area);
	
	// Calculate the interval length 
	timeIntervalLength = (double)(finishTime.tv_sec-startTime.tv_sec)*1000000 
	                   + (double)(finishTime.tv_usec-startTime.tv_usec);
	timeIntervalLength = timeIntervalLength/1000;

	//Print the interval lenght
	if (argc < 2){
		printf("Interval length: %g msec.\n", timeIntervalLength);
	} else { 
		printf("%g\n", timeIntervalLength);
	}

	// Free memory
	hipFree(area);
        
	return 0;
}
