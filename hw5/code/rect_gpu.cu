#include <stdio.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define		NSTEPS	1048576
#define		P_START	0 
#define		P_END	10 

struct timeval startTime;
struct timeval finishTime;
double timeIntervalLength;

__global__ void CUDA_rect(const double h, double* area){
	int my_tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (0 < my_tid && my_tid < NSTEPS)
		atomicAdd(area, cos(P_START+my_tid*h));
}

int main(int argc, char* argv[]){
	// Define variables
	double h;
	double *area;

	// Allocate variables
	hipMallocManaged(&area, sizeof(double));
	
	// Initialize variables
	h = (double)(P_END-P_START)/NSTEPS;
	*area = 0.0;

	// Get the start time
	gettimeofday(&startTime, NULL);

	CUDA_rect<<<1024,1024>>>(h, area);
	hipDeviceSynchronize();
	*area = h*(*area);

	// Get the end time
	gettimeofday(&finishTime, NULL);
	
	// Check results
	printf("Result : %.2lf \n", *area);
	
	// Calculate the interval length 
	timeIntervalLength = (double)(finishTime.tv_sec-startTime.tv_sec) * 1000000 
		     		   + (double)(finishTime.tv_usec-startTime.tv_usec);
	timeIntervalLength = timeIntervalLength/1000;

	// Print the interval lenght
	printf("Interval length: %g msec.\n", timeIntervalLength);

	// Free memory
	hipFree(area);
        
	return 0;
}
