#include <stdio.h>
#include <time.h>
#include <sys/time.h> 
#include <hip/hip_runtime.h>

#define		N	1024

struct timeval startTime;
struct timeval finishTime;
double timeIntervalLength;

__global__ void CUDA_mmult(const double inputArrayA[], const double inputArrayB[], double outputArrayC[]){
	int my_tid = blockDim.x*blockIdx.x + threadIdx.x;
	int i = my_tid/N;
	int j = my_tid%N;
	if (0 < my_tid && my_tid < N*N){
		int k;
		double my_C = 0.0;
		for (k = 0; k < N; k++)
			my_C += inputArrayA[i*N+k]*inputArrayB[k*N+j];
		outputArrayC[i*N+j] = my_C;
	}
}

int main(int argc, char* argv[]){

	// Define variables
	int i, j;
	double totalSum;
	double expectedSum;
	double* inputArrayA;
	double* inputArrayB;
	double* outputArrayC;

	// Allocate arrays
	hipMallocManaged(&inputArrayA, N*N*sizeof(double));
	hipMallocManaged(&inputArrayB, N*N*sizeof(double));
	hipMallocManaged(&outputArrayC, N*N*sizeof(double));
	
	// Initialize arrays
	for (i = 0; i < N; i++){
		for (j = 0; j < N; j++){
			inputArrayA[i*N+j] = i*N+j;
			inputArrayB[i*N+j] = j*(N+1);
			outputArrayC[i*N+j] = 0.0;
		}
	}

	// Get the start time
	gettimeofday(&startTime, NULL);
   
	// Matrix multiplication
	CUDA_mmult<<<1024,1024>>>(inputArrayA, inputArrayB, outputArrayC);
	hipDeviceSynchronize();
	
	// Get the end time
	gettimeofday(&finishTime, NULL);

	// Check results
	totalSum = 0;
	expectedSum = 0;
	for (i = 0; i < N; i++){
		for (j = 0; j < N; j++){
			totalSum += outputArrayC[i*N+j];
			expectedSum += 1.0*(N*N*i + (N*(N-1))/2)*j*(N+1);
		}
	}
	printf("\nTotal Sum = %g\n", totalSum);
	printf("\nExpected Sum = %g\n", expectedSum);

	//Calculate the interval length 
	timeIntervalLength = (double)(finishTime.tv_sec-startTime.tv_sec) * 1000000 
	                   + (double)(finishTime.tv_usec-startTime.tv_usec);
	timeIntervalLength = timeIntervalLength/1000;

	//Print the interval length
	printf("Interval length: %g msec.\n", timeIntervalLength);

	// Free memory
	hipFree(inputArrayA);
	hipFree(inputArrayB);
	hipFree(outputArrayC);

	return 0;  
}
