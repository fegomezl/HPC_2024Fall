#include <stdio.h>
#include <time.h>
#include <sys/time.h> 
#include <hip/hip_runtime.h>

// System size
#define		N	1024

// Structures for profilling
struct timeval startTime;
struct timeval finishTime;
double timeIntervalLength;

// Perform dot product for matrix multiplication on each thread
__global__ void CUDA_mmult(const double inputArrayA[], const double inputArrayB[], double outputArrayC[]){
	int my_tid = blockDim.x*blockIdx.x + threadIdx.x;
	int i = my_tid/N;
	int j = my_tid%N;
	if (my_tid < N*N){
		int k;
		double my_C = 0.0;
		for (k = 0; k < N; k++)
			my_C += inputArrayA[i*N+k]*inputArrayB[k*N+j];
		outputArrayC[my_tid] = my_C;
	}
}

int main(int argc, char* argv[]){

	// Define variables
	int i, j;
	double totalSum;
	double expectedSum;
	double* inputArrayA;
	double* inputArrayB;
	double* outputArrayC;

	// Allocate variables
	hipMallocManaged(&inputArrayA, N*N*sizeof(double));
	hipMallocManaged(&inputArrayB, N*N*sizeof(double));
	hipMallocManaged(&outputArrayC, N*N*sizeof(double));
	
	// Initialize variables
	for (i = 0; i < N; i++){
		for (j = 0; j < N; j++){
			inputArrayA[i*N+j] = i*N+j;
			inputArrayB[i*N+j] = j*(N+1);
			outputArrayC[i*N+j] = 0.0;
		}
	}

	// Get the start time
	gettimeofday(&startTime, NULL);
   
	// Call device function and synchronize
	CUDA_mmult<<<N*N/1024,1024>>>(inputArrayA, inputArrayB, outputArrayC);
	hipDeviceSynchronize();
	
	// Get the end time
	gettimeofday(&finishTime, NULL);

	// Check results
	if (argc < 2){
		totalSum = 0;
		expectedSum = 0;
		for (i = 0; i < N; i++){
			for (j = 0; j < N; j++){
				totalSum += outputArrayC[i*N+j];
				expectedSum += 1.0*(N*N*i + (N*(N-1))/2)*j*(N+1);
			}
		}
		printf("\nTotal Sum = %g\n", totalSum);
		printf("Expected Sum = %g\n", expectedSum);
	}

	//Calculate the interval length 
	timeIntervalLength = (double)(finishTime.tv_sec-startTime.tv_sec)*1000000 
	                   + (double)(finishTime.tv_usec-startTime.tv_usec);
	timeIntervalLength = timeIntervalLength/1000;

	//Print the interval length
	if (argc < 2){
		printf("Interval length: %g msec.\n", timeIntervalLength);
	} else { 
		printf("%g\n", timeIntervalLength);
	}

	// Free memory
	hipFree(inputArrayA);
	hipFree(inputArrayB);
	hipFree(outputArrayC);

	return 0;  
}
